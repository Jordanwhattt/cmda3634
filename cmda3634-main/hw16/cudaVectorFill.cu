
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//This function does not need to be named this, but kernel is added to remind us this process will take place on the GPU
// nvcc -o cudaHelloRank cudeHelloRank.cu
__global__ void fillVectorKernel(int N, double *x) {

    int thread = threadIdx.x;
    int block = blockIdx.x;
    int dim = blockDim.x;   //number of threads per block
    int grid = gridDim.x;   //number of blocks
    
    int n = thread + block*dim; // n in range [0,grid*dim)

    while(n<N) {
        x[n] = n;
        n += grid*dim;
    }

}


int main(int argc, char** argv) {
    
    int G = 4; // 10 thread blocks
    int B = 64; //20 threads per thread-block

    int N = 100000; //pass this number into the function
    double *c_x;
    double *h_x = (double*) calloc(N, sizeof(double));

    hipMalloc(&c_x, N*sizeof(double));

    fillVectorKernel <<< G, B >>> (N, c_x);

    hipMemcpy(h_x, c_x, N*sizeof(double), hipMemcpyDeviceToHost);
    
    int n;
    for(n = 0; n < N;n++) {
        printf("h_x[%d] = %f\n", n, h_x[n]);
    } 
    hipFree(c_x);
    
}
